#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include "../fused_distance_nn.cuh"
 #include <raft/core/kvp.hpp>  // raft::KeyValuePair
 
 #include <cstdint>  // int64_t
 
 #define instantiate_ffanns_distance_fusedDistanceNNMinReduce(DataT, OutT, IdxT) \
   template void ffanns::distance::fusedDistanceNNMinReduce<DataT, OutT, IdxT>(  \
     OutT * min,                                                               \
     const DataT* x,                                                           \
     const DataT* y,                                                           \
     const DataT* xn,                                                          \
     const DataT* yn,                                                          \
     IdxT m,                                                                   \
     IdxT n,                                                                   \
     IdxT k,                                                                   \
     void* workspace,                                                          \
     bool sqrt,                                                                \
     bool initOutBuffer,                                                       \
     bool isRowMajor,                                                          \
     ffanns::distance::DistanceType metric,                                      \
     float metric_arg,                                                         \
     hipStream_t stream)
 
 instantiate_ffanns_distance_fusedDistanceNNMinReduce(float, float, int);
 instantiate_ffanns_distance_fusedDistanceNNMinReduce(float, float, int64_t);
 
 // We can't have comma's in the macro expansion, so we use the COMMA macro:
 #define COMMA ,
 
 instantiate_ffanns_distance_fusedDistanceNNMinReduce(float, raft::KeyValuePair<int COMMA float>, int);
 instantiate_ffanns_distance_fusedDistanceNNMinReduce(float,
                                                    raft::KeyValuePair<int64_t COMMA float>,
                                                    int64_t);
 
 #undef COMMA
 
 #undef instantiate_ffanns_distance_fusedDistanceNNMinReduce
 